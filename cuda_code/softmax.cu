#include "hip/hip_runtime.h"
// matrix row softmax
#define WARP_SIZE 32
// 单block softmax，模板版本
template <class T>
struct MaxOp {
    __device__ __forceinline__ T operator()(const T& x, const T& y) {
        return x > y ? x : y;
    }
};

template <>
struct MaxOp<float> {
    __device__ __forceinline__ float operator()(const float& x, const float& y) {
        return fmaxf(x, y);
    }
};

template <class T>
struct SumOp {
    __device__ __forceinline__ T operator()(const T& x, const T& y) {
        return x + y;
    }
};

template <class Operator, const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_fp32(float val, Operator& op) {
#pragma unroll
    for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
        val = op(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

template <class Operator, const int NUM_THREADS = 256>
__device__ float block_reduce_fp32(float val, int N, Operator& op) {
    int tid = threadIdx.x;
    int warp = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    __shared__ float reduce[NUM_WARPS];
    float value = warp_reduce_fp32<Operator, WARP_SIZE>(val, op);
    if (lane == 0) {
        reduce[warp] = value;
    }
    __syncthreads();
    value = (lane < NUM_WARPS) ? reduce[lane] : 0.0f;
    value = warp_reduce_fp32<Operator, NUM_WARPS>(value, op);
    // value = __shfl_sync(0xffffffff, value, 0,32);
    return value;
}

template <const int NUM_THREADS = 256>
__global__ void safe_softmax_block_fp32(float* x, float* y, int M, int N) {
    int tid = threadIdx.x;
    int row = blockIdx.x;
    if (row >= M)
        return;
    MaxOp<float> max_op;
    SumOp<float> sum_op;
    float max_val_thread = -FLT_MAX;
    for (int i = tid; i < N; i += NUM_THREADS) {
        max_val_thread = fmaxf(max_val_thread, x[row * N + i]);
    }
    float max_val =
        block_reduce_fp32<MaxOp<float>, NUM_THREADS>(max_val_thread, N, max_op);
    float exp_val = 0.0;
    for (int i = tid; i < N; i += NUM_THREADS) {
        exp_val += expf(x[row * N + i] - max_val);
    }
    float exp_sum =
        block_reduce_fp32<SumOp<float>, NUM_THREADS>(exp_val, N, sum_op);
    for (int i = tid; i < N; i += NUM_THREADS) {
        y[row * N + i] = expf(x[row * N + i] - max_val) / exp_sum;
    }
}
